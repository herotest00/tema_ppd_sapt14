#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""


#define PIXEL double

typedef struct _BITMAP{
    int height;
    int width;
    PIXEL* bytes = NULL;
} BITMAP, *PBITMAP;

void destroyBitmap(PBITMAP bitmap) {
    if (bitmap != NULL) {
        if (bitmap->bytes != NULL) {
            hipFree(&(bitmap->bytes));
        }
        hipFree(&bitmap);
    }
}

__device__
PIXEL _max(PIXEL x, PIXEL y) {
    if (x > y) {
        return x;
    }
    return y;
}

__device__
PIXEL _min(PIXEL x, PIXEL y) {
    if (x < y) {
        return x;
    }
    return y;
}

__device__
PIXEL applyFilter(PBITMAP image, PBITMAP kernel, int offset)
{
    int row = offset / image->width;
    int col = offset % image->width;
    PIXEL returned = 0;
    int kernelDimX = kernel->width / 2;
    int kernelDimY = kernel->height / 2;
    int startY = row - kernelDimY;
    int startX = col - kernelDimX;

    for (int i = 0; i <= kernel->height; i++) 
    {
        for (int j = 0; j <= kernel->width; j++)
        {
            int newY = _max(0, _min(startY + i, image->height - 1));
            int newX = _max(0, _min(startX + j, image->width - 1));
            int position = newY * image->width + newX;
            returned += image->bytes[position] * kernel->bytes[i * kernel->width + j];
        }
    }

    return returned;
}

__global__
void gaussian_filter(PBITMAP image, PBITMAP kernel, PBITMAP result) 
{
    int stride = blockDim.x * gridDim.x;
    int rest = (image->width * image->height) % stride, cat = image->width * image->height / stride;
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int start = cat * threadId + _min(threadId, rest);
    int extra = 0;

    if (threadId < (image->width * image->height) % stride) {
        extra = 1;
    }

    for (int i = 0; i < cat + extra && start + i < image->height * image->width; i++) {
        int offset = start + i;
        PIXEL x = applyFilter(image, kernel, offset);
        result->bytes[offset] = x;
    }
}

int readMatrix(PBITMAP& matrix, std::ifstream& fin) {
    int error = hipMallocManaged(&matrix, sizeof(BITMAP));

    if (error != hipSuccess) {
        return -1;
    }
    fin >> matrix->height >> matrix->width;

    int size = matrix->height * matrix->width;
    error = hipMallocManaged(&(matrix->bytes), size * sizeof(PIXEL));
    if (error != hipSuccess) {
        return -2;
    }

    for (int i = 0; i < size; i++) {
        fin >> matrix->bytes[i];
    }

    return 0;
}

int main(int argc, char** argv)
{
    int status = 0;
    std::string filename;
    std::ifstream fin;
    std::ofstream fout;
    PBITMAP image = NULL, kernel = NULL, result = NULL;

    if (argc < 3) {
        std::cout << "Usage: program filename no_threads\n";
        return -1;
    }
    filename = argv[1];

    fin.open(filename);
    if (!fin.is_open()) {
        std::cout << "Couldn't open file " + filename + "\n";
        goto cleanup;
    }

    if ((status = readMatrix(image, fin)) < 0) {
        std::cout << "Error reading image\n";
        goto cleanup;
    }

    if ((status = readMatrix(kernel, fin)) < 0) {
        std::cout << "Error reading kernel\n";
        goto cleanup;
    }

    if (hipMallocManaged(&result, sizeof(PBITMAP)) != hipSuccess) {
        std::cout << "Error allocating result image\n";
        goto cleanup;
    }
    if (hipMallocManaged(&result->bytes, sizeof(PIXEL) * image->height * image->width) != hipSuccess) {
        std::cout << "Error allocating result image\n";
        goto cleanup;
    }
    result->height = image->height;
    result->width = image->width;

    int noThreads = atoi(argv[2]);
    int noBlocks = (image->height * image->width + noThreads - 1) / noThreads;

    const clock_t beginTime = clock();

    gaussian_filter<<<noBlocks, noThreads>>>(image, kernel, result);
    hipDeviceSynchronize();

    std::cout << 1000.0 * (float(clock() - beginTime) / CLOCKS_PER_SEC);

    fout.open("output.txt");
    if (!fout.is_open()) {
        std::cout << "Couldn't open/create output file\n";
        goto cleanup;
    }

    for (int i = 0; i < result->height; i++) {
        for (int j = 0; j < result->width; j++) {
            fout << result->bytes[i * result->width + j] << " ";
        }
        fout << "\n";
    }

    std::cout << "Succes!\n";

    cleanup:
    if (fin.is_open()) {
        fin.close();
    }
    if (fout.is_open()) {
        fout.close();
    }
    destroyBitmap(image);
    destroyBitmap(kernel);
    destroyBitmap(result);

    return status;
}